#include "hip/hip_runtime.h"
// Copyright 2013 Yangqing Jia

#include <algorithm>
#include <cmath>
#include <cstdlib>
#include <cstring>

#include "caffe/common.hpp"
#include "caffe/util/im2col.hpp"

namespace caffe {

template <typename Dtype>
__global__ void im2col_gpu_kernel(const int n, const Dtype* data_im,
    const int height, const int width, const int ksize, const int pad,
    const int stride, const int height_col, const int width_col,
    Dtype* data_col) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n){
      int w_out = index % width_col;
      index /= width_col;
      int h_out = index % height_col;
      int channel_in = index / height_col;
      int channel_out = channel_in * ksize * ksize;
      int h_in = h_out * stride - pad;
      int w_in = w_out * stride - pad;
      data_col += (channel_out * height_col + h_out) * width_col + w_out;
      data_im += (channel_in * height + h_in) * width + w_in;
      for (int i = 0; i < ksize; ++i) {
        for (int j = 0; j < ksize; ++j) {
        	int h = h_in + i;
        	int w = w_in + j;
        	*data_col = (h >= 0 && w >= 0 && h < width && w < height) ?
        	    data_im[i * width + j] : 0;
        	data_col += height_col * width_col;
        }
      }
    }
}

template <typename Dtype>
void im2col_gpu(const Dtype* data_im, const int channels,
    const int height, const int width, const int ksize, const int pad,
    const int stride, Dtype* data_col, const int id) {
  // We are going to launch channels * height_col * width_col kernels, each
  // kernel responsible for copying a single-channel grid.
  int height_col = (height + 2 * pad - ksize) / stride + 1;
  int width_col = (width + 2 * pad - ksize) / stride + 1;
  int num_kernels = channels * height_col * width_col;
  // NOLINT_NEXT_LINE(whitespace/operators)
  im2col_gpu_kernel<Dtype><<<CAFFE_GET_BLOCKS(num_kernels),
                             CAFFE_CUDA_NUM_THREADS, 0,
                             Caffe::stream_handle(id)>>>(
      num_kernels, data_im, height, width, ksize, pad, stride, height_col,
      width_col, data_col);
  CUDA_POST_KERNEL_CHECK;
}


// Explicit instantiation
template void im2col_gpu<float>(const float* data_im, const int channels,
    const int height, const int width, const int ksize, const int pad,
    const int stride, float* data_col, const int id);
template void im2col_gpu<double>(const double* data_im, const int channels,
    const int height, const int width, const int ksize, const int pad,
    const int stride, double* data_col, const int id);

template <typename Dtype>
__global__ void col2im_gpu_kernel(const int n, const Dtype* data_col,
    const int height, const int width, const int channels, const int ksize,
    const int pad, const int stride, const int height_col, const int width_col,
    Dtype* data_im) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n) {
      Dtype val = 0;
      int w = index % width + pad;
      int h = (index / width) % height + pad;
      int c = index / (width * height);
      // compute the start and end of the output
      int w_col_start = (w < ksize) ? 0 : (w - ksize) / stride + 1;
      int w_col_end = min(w / stride + 1, width_col);
      int h_col_start = (h < ksize) ? 0 : (h - ksize) / stride + 1;
      int h_col_end = min(h / stride + 1, height_col);
      /*
      for (int h_col = h_col_start; h_col < h_col_end; ++h_col) {
        for (int w_col = w_col_start; w_col < w_col_end; ++w_col) {
          // the col location: [c * width * height + h_out, w_out]
          int c_col = c * ksize * ksize + (h - h_col * stride) * ksize + (w - w_col * stride);
          val += data_col[(c_col * height_col + h_col) * width_col + w_col];
        }
      }
      */
      // equivalent implementation
      int offset = (c * ksize * ksize + h * ksize + w) * height_col * width_col;
      int coeff_h_col = (1 - stride * ksize * height_col) * width_col;
      int coeff_w_col = (1 - stride * height_col * width_col);
      for (int h_col = h_col_start; h_col < h_col_end; ++h_col) {
        for (int w_col = w_col_start; w_col < w_col_end; ++w_col) {
          val += data_col[offset + h_col * coeff_h_col + w_col * coeff_w_col];
        }
      }
      data_im[index] = val;
    }
}

template <typename Dtype>
void col2im_gpu(const Dtype* data_col, const int channels,
    const int height, const int width, const int ksize, const int pad,
    const int stride, Dtype* data_im, const int id) {
  // CUDA_CHECK(hipMemset(data_im, 0,
  //            sizeof(Dtype) * height * width * channels));
  int height_col = (height + 2 * pad - ksize) / stride + 1;
  int width_col = (width + 2 * pad - ksize) / stride + 1;
  int num_kernels = channels * height * width;
  // To avoid involving atomic operations, we will launch one kernel per
  // bottom dimension, and then in the kernel add up the top dimensions.
  // NOLINT_NEXT_LINE(whitespace/operators)
  col2im_gpu_kernel<Dtype><<<CAFFE_GET_BLOCKS(num_kernels),
                             CAFFE_CUDA_NUM_THREADS, 0, 
                             Caffe::stream_handle(id)>>>(
      num_kernels, data_col, height, width, channels, ksize, pad, stride,
      height_col, width_col, data_im);
  CUDA_POST_KERNEL_CHECK;
}


// Explicit instantiation
template void col2im_gpu<float>(const float* data_col, const int channels,
    const int height, const int width, const int psize, const int pad,
    const int stride, float* data_im, const int id);
template void col2im_gpu<double>(const double* data_col, const int channels,
    const int height, const int width, const int psize, const int pad,
    const int stride, double* data_im, const int id);

template <typename Dtype>
__global__ void im2col_uw_gpu_kernel(const int n, const Dtype* data_im, const int channels,
    const int height, const int width, const int ksize, const int pad,
    const int stride, const int height_col, const int width_col, const int begin, const int end,
    Dtype* data_col) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n){
      int w_out = index % width_col;
      index /= width_col;
      int h_out = index % height_col;
      int channel_in = index / height_col;
      int channel_out = channel_in * ksize * ksize;
      int h_in = (h_out + begin) * stride - pad;
      int w_in = w_out * stride - pad;
      data_col += (channels * ksize * ksize * (h_out * width_col + w_out) + channel_out);
    //  data_col += (channel_out * height_col + h_out) * width_col + w_out;
      data_im += (channel_in * height + h_in) * width + w_in;
      for (int i = 0; i < ksize; ++i) {
        for (int j = 0; j < ksize; ++j) {
        	int h = h_in + i;
        	int w = w_in + j;
        	*data_col = (h >= 0 && w >= 0 && h < width && w < height) ?
        	    data_im[i * width + j] : 0;
          data_col += 1;
   //     	data_col += height_col * width_col;
        }
      }
    }
}

template <typename Dtype>
void im2col_uw_gpu(const Dtype* data_im, const int begin, const int end, const int channels,
    const int height, const int width, const int ksize, const int pad,
    const int stride, Dtype* data_col, const int id) {
  // We are going to launch channels * height_col * width_col kernels, each
  // kernel responsible for copying a single-channel grid.
  int height_col = (height + 2 * pad - ksize) / stride + 1;
  height_col = end-begin;
  int width_col = (width + 2 * pad - ksize) / stride + 1;
  int num_kernels = channels * height_col * width_col;
  // NOLINT_NEXT_LINE(whitespace/operators)
  im2col_uw_gpu_kernel<Dtype><<<CAFFE_GET_BLOCKS(num_kernels),
                             CAFFE_CUDA_NUM_THREADS, 0, 
                             Caffe::stream_handle(id)>>>(
      num_kernels, data_im, channels, height, width, ksize, pad, stride, height_col,
      width_col, begin, end, data_col);
  CUDA_POST_KERNEL_CHECK;
}

// Explicit instantiation
template void im2col_uw_gpu<float>(const float* data_im, const int begin, const int end, const int channels,
    const int height, const int width, const int ksize, const int pad,
    const int stride, float* data_col, int id);
template void im2col_uw_gpu<double>(const double* data_im, const int begin, const int end, const int channels,
    const int height, const int width, const int ksize, const int pad,
    const int stride, double* data_col, int id);

template <typename Dtype>
__global__ void col2im_uw_gpu_kernel(const int n, const Dtype* data_col,
    const int height, const int width, const int channels, const int ksize,
    const int pad, const int stride, const int height_col, const int width_col,
    const int begin, const int end, Dtype* data_im) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n) {
      Dtype val = 0;
     // int height_delta = end-begin;
      int w = index % width + pad;
      int h = (index / width) % height + pad;
      int c = index / (width * height);
      // compute the start and end of the output
      int w_col_start = (w < ksize) ? 0 : (w - ksize) / stride + 1;
      int w_col_end = min(w / stride + 1, width_col);
      int h_col_start = (h < ksize) ? 0 : (h - ksize) / stride + 1;
      int h_col_end = min(h / stride + 1, height_col);
      // equivalent implementation
      int offset = (c * ksize * ksize + h * ksize + w) * height_col * width_col;
      int coeff_h_col = (1 - stride * ksize * height_col) * width_col;
      int coeff_w_col = (1 - stride * height_col * width_col);
      for (int h_col = max(h_col_start,begin); h_col < min(h_col_end, end); ++h_col) {
        for (int w_col = w_col_start; w_col < w_col_end; ++w_col) {
          int pos = offset + h_col * coeff_h_col + w_col * coeff_w_col;
          int w_new = pos % width_col;
          pos/=width_col;
          int h_new = pos % height_col;
          pos/=height_col;
          h_new -= begin;
          pos = (h_new * width_col + w_new) * channels * ksize * ksize + pos;
          //pos = (pos * height_delta + h_new) * width_col + w_new; 
          val += data_col[pos];
        }
      }
      data_im[index] += val;
    }
}

template <typename Dtype>
void col2im_uw_gpu(const Dtype* data_col, const int begin, const int end, const int channels,
    const int height, const int width, const int ksize, const int pad,
    const int stride, Dtype* data_im, const int id) {
  // CUDA_CHECK(hipMemset(data_im, 0,
  //            sizeof(Dtype) * height * width * channels));
  int height_col = (height + 2 * pad - ksize) / stride + 1;
  int width_col = (width + 2 * pad - ksize) / stride + 1;
  int num_kernels = channels * height * width;
  // To avoid involving atomic operations, we will launch one kernel per
  // bottom dimension, and then in the kernel add up the top dimensions.
  // NOLINT_NEXT_LINE(whitespace/operators)
  col2im_uw_gpu_kernel<Dtype><<<CAFFE_GET_BLOCKS(num_kernels),
                             CAFFE_CUDA_NUM_THREADS, 0, 
                             Caffe::stream_handle(id)>>>(
      num_kernels, data_col, height, width, channels, ksize, pad, stride,
      height_col, width_col, begin, end, data_im);
  CUDA_POST_KERNEL_CHECK;
}


// Explicit instantiation
template void col2im_uw_gpu<float>(const float* data_col, const int begin, const int end, const int channels,
    const int height, const int width, const int psize, const int pad,
    const int stride, float* data_im, int id);
template void col2im_uw_gpu<double>(const double* data_col, const int begin, const int end, const int channels,
    const int height, const int width, const int psize, const int pad,
    const int stride, double* data_im, int id);

}  // namespace caffe
